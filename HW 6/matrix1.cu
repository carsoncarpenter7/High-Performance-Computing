#include <stdio.h>
#include <hip/hip_runtime.h>

//  nvcc matrix.cu
// ./a.out

// From Prob 1 (keep to intitilize matrix)
__global__ void dkernel(unsigned *orig_matrix, int *size) {
    // initialize matrix
    unsigned id = threadIdx.x * blockDim.y + threadIdx.y;
    orig_matrix[id] = id;

    // if((orig_matrix[id] % 2) == 0) {
    //     count++;
    // }
}

// For Prob 2
__global__ void kernal ( unsigned *orig_matrix, unsigned *result, unsigned size) {
    // Compute square kernal
    unsigned id = blockIdx.x * blockDim.x + threadIdx.x;
    for (unsigned i = 0; i < size; ++i) 
    {
        for (unsigned ii = 0; ii < size; ++ii) 
        {
            // multiply/combine original matrixes
            result[id * size + i] += orig_matrix[id * size + ii] *
            orig_matrix[ii * size + i];
        }
    }
}

// 8x8 matrix (NxN)
#define N 8
#define M 8

int main() {

    dim3 block(N, M, 1);
    unsigned *orig_matrix, *new_matrix, *result, *squared;
    int *size;

    hipMalloc(&orig_matrix, N * M * sizeof(unsigned));
    hipMalloc(&size, sizeof(int));
    new_matrix = (unsigned *)malloc(N * M * sizeof(unsigned));

    //Kernel 1: For Prob 1
    dkernel<<<1, block>>>(orig_matrix, size);

    // ================ HW 5 EX: ================ 
    // cudaMemcpy (newArray, array, size * sizeof(unsigned), cudaMemcpyDeviceToHost);
    // ========================================== 

     // send to host
    hipMemcpy(new_matrix, orig_matrix, N * M * sizeof(unsigned), hipMemcpyDeviceToHost);
    
    // Not sure if needed? count not working
    //cudaMemcpy(&count, size, sizeof(int), cudaMemcpyDeviceToHost);
    
    // For Prob 2
    squared = (unsigned *)malloc(N * M * sizeof(unsigned));
    hipMalloc(&result, N * M * sizeof(unsigned));

    //Kernel 2: For Prob 2
    kernal<<<1, N>>>(orig_matrix, result, N);

    // send to host
    hipMemcpy(squared, result, N * M * sizeof(unsigned), hipMemcpyDeviceToHost);

    //Printed Matrix 1
    printf("Compute Square:\n");
    for (unsigned i = 0; i < N; ++i) 
    {
        for (unsigned ii = 0; ii < N; ++ii) 
        { 
            printf("%2d ", squared[i * N + ii]); 
        } 
        printf("\n"); 
    } 

    return 0;
}