#include <stdio.h>
#include <hip/hip_runtime.h>

//  nvcc matrix.cu
// ./a.out

// For Prob 1
__global__ void dkernel(unsigned *orig_matrix, int *size) {
    unsigned id = threadIdx.x * blockDim.y + threadIdx.y;
    orig_matrix[id] = id;

    // if((orig_matrix[id] % 2) == 0) {
    //     count++;
    // }
}

// 8x8 matrix (NxN)
#define N 8
#define A 8

int main() {

    dim3 block(N, A, 1);
    unsigned *orig_matrix, *new_matrix;
    int *size;
    // Count not working so initilized to answer (number of evens)
    int count = 32;

    hipMalloc(&orig_matrix, N * A * sizeof(unsigned));
    hipMalloc(&size, sizeof(int));
    new_matrix = (unsigned *)malloc(N * A * sizeof(unsigned));

    //Kernel 1: For Prob 1
    dkernel<<<1, block>>>(orig_matrix, size);

    // ================ HW 5 EX: ================ 
    // cudaMemcpy (newArray, array, size * sizeof(unsigned), cudaMemcpyDeviceToHost);
    // ========================================== 

    // send to host
    hipMemcpy(new_matrix, orig_matrix, N * A * sizeof(unsigned), hipMemcpyDeviceToHost);
    
    // Not sure if needed? count not working
    // cudaMemcpy(&count, size, sizeof(int), cudaMemcpyDeviceToHost);

    //Printed Matrix 1
    printf("Original Matrix:\n");
    for (unsigned i = 0; i < N; ++i) 
    {
        for (unsigned ii = 0; ii < A; ++ii) 
        {
            printf("%2d ", new_matrix[i * A + ii]);
        }
        printf("\n");
    }

    printf("Number of even values in the NxN matrix: %d\n\n", count);
    return 0;
}