#include <stdio.h>
#include <hip/hip_runtime.h>

// For Prob 1
__global__ void dkernel(unsigned *array, int size)
{
    unsigned id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < size)
    {
        // For Prob 1 
        array[id] = 0;
    }
}

// For Prob 3
__global__ void kernel(unsigned *array, int size)
{
    unsigned id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < size)
    {
        // For Prob 3
        array[id] = id;
    }
}

// Dont change??
#define BLOCKSIZE 1024

int main(void) {
    // For Prob 1 
    unsigned size = 32;

    // For Prob 2/3
    // unsigned size = 1024

    // For Prob 4
    // unsigned size = 8000;

    unsigned *array, *newArray;
    hipMalloc(&array, size * sizeof(unsigned));

    newArray = (unsigned *)malloc(size * sizeof(unsigned));
    unsigned nblocks = ceil((float) size / BLOCKSIZE);

    printf ("This is an array of size %d and filled with 0's. \n", size);

    //Kernel 1: For Prob 1/2
    dkernel<<<nblocks, BLOCKSIZE>>> (array, size);

    // Kernel 2: For Prob 3/4
    // kernel<<<nblocks, BLOCKSIZE>>> (array, size);

    hipMemcpy (newArray, array, size * sizeof(unsigned), hipMemcpyDeviceToHost);
    
    // Print Array
    for (unsigned ai = 0; ai < size; ++ai) 
    {
        printf("%4d \n", newArray[ai]);
    }
}